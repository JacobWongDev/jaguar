#include <stdlib.h>
#include <float.h>
#include "cosq.h"
#include "spdlog/spdlog.h"
#include "cuda/nnc.cuh"
#include "cuda/cc.cuh"
#include "cuda/dist.cuh"
#include "cuda/nvidia.cuh"
#include "ext.h"

Split::Split(COSQ* cosq, Device* device) {
  this->cosq = cosq;
  this->device = device;
}

void Split::split_lt32() {
  double* temp = NULL;
  double* s_codebook = (double*) malloc(sizeof(double) * cosq->levels);
  cosq->q_points = (double*) malloc(sizeof(double) * cosq->levels);
  // Compute centroid of training sequence
  double sum = 0;
  for(int i = 0; i < cosq->training_size; i++)
      sum += cosq->training_sequence[i];
  cosq->q_points[0] = sum / cosq->training_size;
  nnc_block_size = {WARP_SIZE, 1, 1};
  cc_cell_sums = (double*) malloc(sizeof(double) * cosq->levels);
  cc_cardinality = (unsigned int*) malloc(sizeof(unsigned int) * cosq->levels);
  unsigned int rate = 0;
  unsigned int s_levels = 1;
  while(s_levels < cosq->levels) {
    // printArr(cosq->q_points, s_levels);
    for(int i = 0; i < s_levels; i++) {
      s_codebook[2*i] = cosq->q_points[i] - delta;
      s_codebook[2*i+1] = cosq->q_points[i] + delta;
    }
    temp = cosq->q_points;
    cosq->q_points = s_codebook;
    s_codebook = temp;
    s_levels <<= 1;
    rate++;
    checkCudaErrors(hipMemset(device->cc_cardinality, 0, s_levels*sizeof(unsigned int)));
    checkCudaErrors(hipMemset(device->cc_cell_sums, 0, s_levels*sizeof(double)));
    nnc_grid_size = {cosq->training_size * s_levels / WARP_SIZE, 1, 1};
    checkCudaErrors(hipMemcpy(device->q_points, cosq->q_points, sizeof(double) * s_levels, hipMemcpyHostToDevice));
    compute_error_matrix(cosq->error_matrix, s_levels, rate);
    checkCudaErrors(hipMemcpy(device->error_matrix, cosq->error_matrix, sizeof(double) * s_levels * s_levels, hipMemcpyHostToDevice));
    s_nnc_lt32<<<nnc_grid_size, nnc_block_size>>>(s_levels, device->training_sequence, device->q_points,
        device->error_matrix, device->cc_cell_sums, device->cc_cardinality);
    checkCudaErrors(hipMemcpy(cc_cell_sums, device->cc_cell_sums, sizeof(double) * s_levels, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(cc_cardinality, device->cc_cardinality, sizeof(unsigned int) * s_levels, hipMemcpyDeviceToHost));
    cc_lt32(s_levels, cosq->error_matrix, cc_cell_sums, cc_cardinality, cosq->q_points);
  }
  free(s_codebook);
  free(cc_cell_sums);
  free(cc_cardinality);
}

void Split::split_ge32() {
  double* temp = NULL;
  double* s_codebook = (double*) malloc(sizeof(double) * cosq->levels);
  cosq->q_points = (double*) malloc(sizeof(double) * cosq->levels);
  cc_cell_sums = (double*) malloc(sizeof(double) * cosq->levels);
  cc_cardinality = (unsigned int*) malloc(sizeof(unsigned int) * cosq->levels);
  // Compute centroid of training sequence
  double sum = 0;
  for(int i = 0; i < cosq->training_size; i++)
    sum += cosq->training_sequence[i];
  cosq->q_points[0] = sum / cosq->training_size;
  nnc_block_size = {WARP_SIZE, 1, 1};
  cc_block_size = {WARP_SIZE, 1, 1};
  unsigned int rate = 0;
  unsigned int s_levels = 1;
  while(s_levels < 32) {
    for(int i = 0; i < s_levels; i++) {
      s_codebook[2*i] = cosq->q_points[i] - delta;
      s_codebook[2*i+1] = cosq->q_points[i] + delta;
    }
    temp = cosq->q_points;
    cosq->q_points = s_codebook;
    s_codebook = temp;
    s_levels <<= 1;
    rate++;
    checkCudaErrors(hipMemset(device->cc_cardinality, 0, s_levels*sizeof(unsigned int)));
    checkCudaErrors(hipMemset(device->cc_cell_sums, 0, s_levels*sizeof(double)));
    nnc_grid_size = {cosq->training_size * s_levels / WARP_SIZE, 1, 1};
    checkCudaErrors(hipMemcpy(device->q_points, cosq->q_points, sizeof(double) * s_levels, hipMemcpyHostToDevice));
    compute_error_matrix(cosq->error_matrix, s_levels, rate);
    checkCudaErrors(hipMemcpy(device->error_matrix, cosq->error_matrix, sizeof(double) * s_levels * s_levels, hipMemcpyHostToDevice));
    s_nnc_lt32<<<nnc_grid_size, nnc_block_size>>>(s_levels, device->training_sequence, device->q_points,
        device->error_matrix, device->cc_cell_sums, device->cc_cardinality);
    checkCudaErrors(hipMemcpy(cc_cell_sums, device->cc_cell_sums, sizeof(double) * s_levels, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(cc_cardinality, device->cc_cardinality, sizeof(unsigned int) * s_levels, hipMemcpyDeviceToHost));
    cc_lt32(s_levels, cosq->error_matrix, cc_cell_sums, cc_cardinality, cosq->q_points);
  }
  nnc_grid_size = {cosq->training_size, 1, 1};
  while(s_levels < cosq->levels) {
    for(int i = 0; i < s_levels; i++) {
      s_codebook[2*i] = cosq->q_points[i] - delta;
      s_codebook[2*i+1] = cosq->q_points[i] + delta;
    }
    temp = cosq->q_points;
    cosq->q_points = s_codebook;
    s_codebook = temp;
    s_levels <<= 1;
    rate++;
    checkCudaErrors(hipMemset(device->cc_cardinality, 0, s_levels*sizeof(unsigned int)));
    checkCudaErrors(hipMemset(device->cc_cell_sums, 0, s_levels*sizeof(double)));
    nnc_smem_size = 2 * s_levels * sizeof(double);
    checkCudaErrors(hipMemcpy(device->q_points, cosq->q_points, sizeof(double) * s_levels, hipMemcpyHostToDevice));
    compute_error_matrix(cosq->error_matrix, s_levels, rate);
    checkCudaErrors(hipMemcpy(device->error_matrix, cosq->error_matrix, sizeof(double) * s_levels * s_levels, hipMemcpyHostToDevice));
    s_nnc_ge32<<<nnc_grid_size, nnc_block_size, nnc_smem_size>>>(s_levels, device->training_sequence,
        device->q_points, device->error_matrix, device->cc_cell_sums, device->cc_cardinality);
    cc_grid_size = {s_levels, 1, 1};
    cc_ge32<<<cc_grid_size, cc_block_size>>>(s_levels, device->q_points, device->error_matrix,
        device->cc_cell_sums, device->cc_cardinality);
    checkCudaErrors(hipMemcpy(cosq->q_points, device->q_points, sizeof(double) * s_levels, hipMemcpyDeviceToHost));
  }
  free(s_codebook);
  free(cc_cell_sums);
  free(cc_cardinality);
}

/**
 * Allocate memory for device arrays.
 */
Device::Device(COSQ* cosq) {
  // Memory allocation
  checkCudaErrors(hipMalloc((void **) &training_sequence, (cosq->training_size)*sizeof(double)));
  checkCudaErrors(hipMalloc((void **) &error_matrix, (cosq->levels)*(cosq->levels)*sizeof(double)));
  checkCudaErrors(hipMalloc((void **) &q_points, (cosq->levels)*sizeof(double)));

  checkCudaErrors(hipMalloc((void **) &q_cells, (cosq->training_size)*sizeof(unsigned int)));

  checkCudaErrors(hipMalloc((void **) &cc_cardinality, (cosq->levels)*sizeof(unsigned int)));
  checkCudaErrors(hipMalloc((void **) &cc_cell_sums, (cosq->levels)*sizeof(double)));

  checkCudaErrors(hipMalloc((void **) &reduction_sums, (cosq->training_size)*sizeof(double)));

  // Memory copying
  checkCudaErrors(hipMemcpy(training_sequence, cosq->training_sequence,
                            (cosq->training_size)*sizeof(double), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemset(cc_cardinality, 0, (cosq->levels)*sizeof(unsigned int)));
  checkCudaErrors(hipMemset(cc_cell_sums, 0, (cosq->levels)*sizeof(double)));

  // CUDA kernel launch params
  nnc_ge32_grid_size = {cosq->training_size, 1, 1};
  nnc_ge32_block_size = {WARP_SIZE, 1, 1};
  nnc_lt32_grid_size = {cosq->training_size * cosq->levels / WARP_SIZE, 1, 1};
  nnc_lt32_block_size = {WARP_SIZE, 1, 1};
  nnc_smem_size = 2 * cosq->levels * sizeof(double);

  cc_grid_size = {cosq->levels, 1, 1};
  cc_block_size = {WARP_SIZE, 1, 1};

  dist_grid_size = {cosq->training_size / WARP_SIZE, 1, 1};
  dist_block_size = {WARP_SIZE, 1, 1};
  dist_smem_size = cosq->levels * sizeof(double);
}

/**
 * Free all memory on device.
 */
Device::~Device() {
  checkCudaErrors(hipFree(training_sequence));
  checkCudaErrors(hipFree(error_matrix));
  checkCudaErrors(hipFree(q_points));
  checkCudaErrors(hipFree(q_cells));
  checkCudaErrors(hipFree(cc_cardinality));
  checkCudaErrors(hipFree(cc_cell_sums));
  checkCudaErrors(hipFree(reduction_sums));
}

/**
 *
 */
COSQ::COSQ(double* training_sequence, const unsigned int* training_size, const unsigned int* bit_rate) {
  this->bit_rate = *bit_rate;
  this->levels = 1 << *bit_rate;
  this->training_sequence = training_sequence;
  this->training_size = *training_size;
  this->error_matrix = (double*) malloc(levels*levels*sizeof(double));
  if(COSQ::error_matrix == nullptr) {
    spdlog::error("Memory Allocation error: Failed to allocate memory for error_matrix!");
    return;
  }
  COSQ::q_points = (double*) malloc(levels*sizeof(double));
  if(COSQ::q_points == nullptr) {
    spdlog::error("Memory Allocation error: Failed to allocate memory for q_points!");
    return;
  }
  device = new Device(this);
}

COSQ::~COSQ() {
  free(error_matrix);
  free(q_points);
  delete device;
}

/**
 *
 */
inline double polya_urn_error(int j, int i, int num_bits) {
  double temp;
  int x = j ^ i;
  int previous;
  if(x & 1 == 1) {
    temp = POLYA_EPSILON;
    previous = 1;
  } else {
    temp = 1 - POLYA_EPSILON;
    previous = 0;
  }
  x >>= 1;
  for(int i = 1; i < num_bits; i++) {
    if(x & 1 == 1) {
      temp *= (POLYA_EPSILON + previous * POLYA_DELTA) / (1 + POLYA_DELTA);
      previous = 1;
    } else {
      temp *= ((1 - POLYA_EPSILON) + (1 - previous)*POLYA_DELTA) / (1 + POLYA_DELTA);
      previous = 0;
    }
    x >>= 1;
  }
  return temp;
}

/**
 * TODO: Use CUDA to accelerate this
 */
void compute_error_matrix(double* error_matrix, unsigned int levels, unsigned int bit_rate) {
  for(int i = 0; i < levels; i++) {
    for(int j = 0; j < levels; j++) {
      error_matrix[j + i * levels] = polya_urn_error(j, i, bit_rate);
    }
  }
}

void cc_lt32(unsigned int levels, double* error_matrix, double* cc_sums,
    unsigned int* cc_cardinality, double* q_points) {
  double numerator = 0;
  double denominator = 0;
  for (int j = 0; j < levels; j++) {
    for (int i = 0; i < levels; i++) {
        numerator += error_matrix[j + levels * i] * cc_sums[i];
    }
    for (int i = 0; i < levels; i++) {
        denominator += error_matrix[j + levels * i] * cc_cardinality[i];
    }
    q_points[j] = numerator / denominator;
    numerator = 0;
    denominator = 0;
  }
}

/**
 *
 */
double* COSQ::cosq_lt32() {
  double dist_prev = DBL_MAX, dist_curr = 0;
  Split split(this, device);
  split.split_lt32();
  checkCudaErrors(hipMemcpy(device->q_points, q_points, levels * sizeof(double), hipMemcpyHostToDevice));
  compute_error_matrix(error_matrix, levels, bit_rate);
  checkCudaErrors(hipMemcpy(device->error_matrix, error_matrix, levels * levels * sizeof(double), hipMemcpyHostToDevice));
  // For sequential CC
  double* cc_sums_lt32 = (double*) malloc(sizeof(double) * levels);
  unsigned int* cc_cardinal_lt32 = (unsigned int*) malloc(sizeof(unsigned int) * levels);
  // COSQ algorithm
  while(true) {
    checkCudaErrors(hipMemset(device->cc_cardinality, 0, levels*sizeof(unsigned int)));
    checkCudaErrors(hipMemset(device->cc_cell_sums, 0, levels*sizeof(double)));
    nnc_lt32<<<device->nnc_lt32_grid_size, device->nnc_lt32_block_size>>>(levels, device->training_sequence, device->q_points,
        device->error_matrix, device->q_cells, device->cc_cell_sums, device->cc_cardinality);
    checkCudaErrors(hipMemcpy(cc_sums_lt32, device->cc_cell_sums, sizeof(double) * levels, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(cc_cardinal_lt32, device->cc_cardinality, sizeof(unsigned int) * levels, hipMemcpyDeviceToHost));
    cc_lt32(levels, error_matrix, cc_sums_lt32, cc_cardinal_lt32, q_points);
    checkCudaErrors(hipMemcpy(device->q_points, q_points, sizeof(double) * levels, hipMemcpyHostToDevice));
    distortion_gather_lt32<<<device->dist_grid_size, device->dist_block_size>>>(levels, device->training_sequence,
        device->q_points, device->error_matrix, device->q_cells, device->reduction_sums);
    dist_curr = distortion_reduce(training_size, device->reduction_sums);
    spdlog::info("Distortion is {:f}", dist_curr);
    if((dist_prev - dist_curr) / dist_prev < THRESHOLD) {
      break;
    }
    dist_prev = dist_curr;
  }
  // TODO: Return copy of COSQ::q_points
  return nullptr;
  free(cc_sums_lt32);
  free(cc_cardinal_lt32);
}

/**
 *
 */
double* COSQ::cosq_ge32() {
  double dist_prev = DBL_MAX, dist_curr = 0;
  Split split(this, device);
  split.split_ge32();
  checkCudaErrors(hipMemcpy(device->q_points, q_points, levels * sizeof(double), hipMemcpyHostToDevice));
  compute_error_matrix(error_matrix, levels, bit_rate);
  checkCudaErrors(hipMemcpy(device->error_matrix, error_matrix, levels * levels * sizeof(double), hipMemcpyHostToDevice));
  // COSQ algorithm
  while(true) {
    checkCudaErrors(hipMemset(device->cc_cardinality, 0, levels*sizeof(unsigned int)));
    checkCudaErrors(hipMemset(device->cc_cell_sums, 0, levels*sizeof(double)));
    nnc_ge32<<<device->nnc_ge32_grid_size, device->nnc_ge32_block_size, device->nnc_smem_size>>>(levels, device->training_sequence,
        device->q_points, device->error_matrix, device->q_cells, device->cc_cell_sums, device->cc_cardinality);
    cc_ge32<<<device->cc_grid_size, device->cc_block_size>>>(levels, device->q_points, device->error_matrix,
        device->cc_cell_sums, device->cc_cardinality);
    distortion_gather_ge32<<<device->dist_grid_size, device->dist_block_size, device->dist_smem_size>>>(levels, device->training_sequence,
        device->q_points, device->error_matrix, device->q_cells, device->reduction_sums);
    dist_curr = distortion_reduce(training_size, device->reduction_sums);
    spdlog::info("Distortion is {:f}", dist_curr);
    if((dist_prev - dist_curr) / dist_prev < THRESHOLD) {
      break;
    }
    dist_prev = dist_curr;
  }
  // TODO: Return copy of COSQ::q_points
  return nullptr;
}

/**
 *
 */
double* COSQ::train() {
  if(training_sequence == nullptr || training_size == 0) {
    spdlog::error("Failed to train COSQ: Invalid training sequence or size!");
    return nullptr;
  }
  if(levels >= 32) {
    return cosq_ge32();
  } else {
    return cosq_lt32();
  }
}
