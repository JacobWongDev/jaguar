#include "cosq.h"
#include <stdlib.h>
#include <float.h>
#include "spdlog/spdlog.h"
#include "cuda/nnc.cuh"
#include "cuda/cc.cuh"
#include "cuda/dist.cuh"
#include "cuda/nvidia.cuh"
#include "ext.h"

/**
 * Host variables
 */
double* COSQ::training_sequence;
unsigned int COSQ::training_size;
unsigned int COSQ::levels;
unsigned int COSQ::bit_rate;
double* COSQ::error_matrix;
double* COSQ::q_points;


/**
 * Device variables
 */
// General
double* COSQ::Device::training_sequence;
double* COSQ::Device::error_matrix;
double* COSQ::Device::q_points;
// NNC
dim3 COSQ::Device::nnc_ge32_grid_size;
dim3 COSQ::Device::nnc_ge32_block_size;
dim3 COSQ::Device::nnc_lt32_grid_size;
dim3 COSQ::Device::nnc_lt32_block_size;
unsigned int COSQ::Device::nnc_smem_size;
unsigned int* COSQ::Device::q_cells;
// CC
dim3 COSQ::Device::cc_grid_size;
dim3 COSQ::Device::cc_block_size;
unsigned int* COSQ::Device::cc_cardinality;
double* COSQ::Device::cc_cell_sums;
// Distortion
dim3 COSQ::Device::dist_grid_size;
dim3 COSQ::Device::dist_block_size;
unsigned int COSQ::Device::dist_smem_size;
double* COSQ::Device::reduction_sums;


/**
 * Allocate memory for device arrays.
 */
void COSQ::Device::init(double* training_sequence_, const unsigned int* training_size, double* error_matrix_, const unsigned int* levels) {
  // Memory allocation
  checkCudaErrors(hipMalloc((void **) &training_sequence, (*training_size)*sizeof(double)));
  checkCudaErrors(hipMalloc((void **) &error_matrix, (*levels)*(*levels)*sizeof(double)));
  checkCudaErrors(hipMalloc((void **) &q_points, (*levels)*sizeof(double)));

  checkCudaErrors(hipMalloc((void **) &q_cells, (*training_size)*sizeof(unsigned int)));

  checkCudaErrors(hipMalloc((void **) &cc_cardinality, (*levels)*sizeof(unsigned int)));
  checkCudaErrors(hipMalloc((void **) &cc_cell_sums, (*levels)*sizeof(double)));

  checkCudaErrors(hipMalloc((void **) &reduction_sums, (*training_size)*sizeof(double)));

  // Memory copying
  checkCudaErrors(hipMemcpy(training_sequence, training_sequence_,
                            (*training_size)*sizeof(double), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(error_matrix, error_matrix_,
                          (*levels)*(*levels)*sizeof(double), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemset(cc_cardinality, 0, (*levels)*sizeof(unsigned int)));
  checkCudaErrors(hipMemset(cc_cell_sums, 0, (*levels)*sizeof(double)));

  // CUDA kernel launch params
  nnc_ge32_grid_size = {*training_size, 1, 1};
  nnc_ge32_block_size = {WARP_SIZE, 1, 1};
  nnc_lt32_grid_size = {*training_size * COSQ::levels / WARP_SIZE, 1, 1};
  nnc_lt32_block_size = {WARP_SIZE, 1, 1};
  nnc_smem_size = 2 * (*levels) * sizeof(double);

  cc_grid_size = {*levels, 1, 1};
  cc_block_size = {WARP_SIZE, 1, 1};

  dist_grid_size = {*training_size / WARP_SIZE, 1, 1};
  dist_block_size = {WARP_SIZE, 1, 1};
  dist_smem_size = (*levels) * sizeof(double);
}

/**
 *
 */
void COSQ::init(double* training_sequence_, const unsigned int* training_size_) {
  COSQ::training_sequence = training_sequence_;
  COSQ::training_size = *training_size_;
  COSQ::error_matrix = (double*) malloc((COSQ::levels)*(COSQ::levels)*sizeof(double));
  if(COSQ::error_matrix == nullptr) {
    spdlog::error("Memory Allocation error: Failed to allocate memory for error_matrix!");
    return;
  }
  compute_error_matrix();
  COSQ::q_points = (double*) malloc((COSQ::levels)*sizeof(double));
  if(COSQ::q_points == nullptr) {
    spdlog::error("Memory Allocation error: Failed to allocate memory for q_points!");
    return;
  }
  Device::init(training_sequence_, training_size_, error_matrix, &levels);
}

void COSQ::finish() {
  free(COSQ::error_matrix);
  Device::finish();
}


/**
 * Free all memory on device.
 */
void COSQ::Device::finish() {
  checkCudaErrors(hipFree(training_sequence));
  checkCudaErrors(hipFree(error_matrix));
  checkCudaErrors(hipFree(q_points));
  checkCudaErrors(hipFree(q_cells));
  checkCudaErrors(hipFree(cc_cardinality));
  checkCudaErrors(hipFree(cc_cell_sums));
  checkCudaErrors(hipFree(reduction_sums));
}



/**
 *
 */
inline double COSQ::polya_urn_error(int j, int i, int num_bits) {
  double temp;
  int x = j ^ i;
  int previous;
  if(x & 1 == 1) {
    temp = POLYA_EPSILON;
    previous = 1;
  } else {
    temp = 1 - POLYA_EPSILON;
    previous = 0;
  }
  x >>= 1;
  for(int i = 1; i < num_bits; i++) {
    if(x & 1 == 1) {
      temp *= (POLYA_EPSILON + previous * POLYA_DELTA) / (1 + POLYA_DELTA);
      previous = 1;
    } else {
      temp *= ((1 - POLYA_EPSILON) + (1 - previous)*POLYA_DELTA) / (1 + POLYA_DELTA);
      previous = 0;
    }
    x >>= 1;
  }
  return temp;
}

/**
 * TODO: Use CUDA to accelerate this
 */
void COSQ::compute_error_matrix() {
  for(int i = 0; i < (levels); i++) {
    for(int j = 0; j < (levels); j++) {
      error_matrix[j + i * (levels)] = polya_urn_error(j, i, bit_rate);
    }
  }
}

// /**
//  * TODO
//  */
// void COSQ::split() {
//   const double delta = 0.01;
//   double* temp = (double*) malloc(sizeof(double) * (levels));
//   free(temp);
// }

void COSQ::Device::nnc(unsigned int* levels) {
  if(*levels >= WARP_SIZE) {
    nnc_ge32<<<COSQ::Device::nnc_ge32_grid_size, COSQ::Device::nnc_ge32_block_size, COSQ::Device::nnc_smem_size>>>(*levels, COSQ::Device::training_sequence,
        COSQ::Device::q_points, COSQ::Device::error_matrix, COSQ::Device::q_cells, COSQ::Device::cc_cell_sums, COSQ::Device::cc_cardinality);
  } else {
    nnc_lt32<<<COSQ::Device::nnc_lt32_grid_size, COSQ::Device::nnc_lt32_block_size>>>(*levels, COSQ::Device::training_sequence, COSQ::Device::q_points,
        COSQ::Device::error_matrix, COSQ::Device::q_cells, COSQ::Device::cc_cell_sums, COSQ::Device::cc_cardinality);
  }
}

void COSQ::cc_lt32(int levels, double* error_matrix, double* cc_sums, unsigned int* cc_cardinality, double* codebook) {
  double numerator = 0;
  double denominator = 0;
  for (int j = 0; j < levels; j++) {
    for (int i = 0; i < levels; i++) {
        numerator += error_matrix[j + levels * i] * cc_sums[i];
    }
    for (int i = 0; i < levels; i++) {
        denominator += error_matrix[j + levels * i] * cc_cardinality[i];
    }
    codebook[j] = numerator / denominator;
    numerator = 0;
    denominator = 0;
  }
}

void COSQ::cc(unsigned int* levels, double* cc_sums_lt32, unsigned int* cc_cardinal_lt32) {
  if(*levels >= WARP_SIZE) {
    cc_ge32<<<COSQ::Device::cc_grid_size, COSQ::Device::cc_block_size>>>(*levels, COSQ::Device::q_points, COSQ::Device::error_matrix,
        COSQ::Device::cc_cell_sums, COSQ::Device::cc_cardinality);
  } else {
    checkCudaErrors(hipMemcpy(cc_sums_lt32, COSQ::Device::cc_cell_sums, sizeof(double) * *levels, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(cc_cardinal_lt32, COSQ::Device::cc_cardinality, sizeof(unsigned int) * *levels, hipMemcpyDeviceToHost));
    cc_lt32(*levels, COSQ::error_matrix, cc_sums_lt32, cc_cardinal_lt32, COSQ::q_points);
    checkCudaErrors(hipMemcpy(COSQ::Device::q_points, COSQ::q_points, sizeof(double) * *levels, hipMemcpyHostToDevice));
  }
}

double COSQ::Device::distortion(unsigned int* levels) {
  if(*levels >= WARP_SIZE) {
    distortion_gather_ge32<<<COSQ::Device::dist_grid_size, COSQ::Device::dist_block_size, COSQ::Device::dist_smem_size>>>(*levels, COSQ::Device::training_sequence,
        COSQ::Device::q_points, COSQ::Device::error_matrix, COSQ::Device::q_cells, COSQ::Device::reduction_sums);
  } else {
    distortion_gather_lt32<<<COSQ::Device::dist_grid_size, COSQ::Device::dist_block_size>>>(*levels, COSQ::Device::training_sequence,
        COSQ::Device::q_points, COSQ::Device::error_matrix, COSQ::Device::q_cells, COSQ::Device::reduction_sums);
  }
  return distortion_reduce(COSQ::training_size, COSQ::Device::reduction_sums);
}

/**
 *
 */
double* COSQ::train(double* training_sequence, const unsigned int* training_size, const unsigned int* bit_rate) {
  double dist_prev = DBL_MAX, dist_curr = 0;
  COSQ::levels = 1 << *bit_rate;
  COSQ::bit_rate = *bit_rate;
  init(training_sequence, training_size);
  // sim_annealing(Host::q_points, training_sequence, training_size, error_matrix, bit_rate, &levels);
  // For now, just use first few training seq elements
  for(int i = 0; i < levels; i++)
    q_points[i] = training_sequence[i];
  checkCudaErrors(hipMemcpy(COSQ::Device::q_points, COSQ::q_points, levels * sizeof(double), hipMemcpyHostToDevice));
  // For sequential CC
  double* cc_sums_lt32 = (double*) malloc(sizeof(double) * levels);
  unsigned int* cc_cardinal_lt32 = (unsigned int*) malloc(sizeof(unsigned int) * levels);
  // COSQ algorithm
  while(true) {
    Device::nnc(&levels);
    COSQ::cc(&levels, cc_sums_lt32, cc_cardinal_lt32);
    dist_curr = Device::distortion(&levels);
    spdlog::info("Distortion is {:f}", dist_curr);
    if((dist_prev - dist_curr) / dist_prev < THRESHOLD) {
      break;
    }
    dist_prev = dist_curr;
    checkCudaErrors(hipMemset(COSQ::Device::cc_cardinality, 0, (levels)*sizeof(unsigned int)));
    checkCudaErrors(hipMemset(COSQ::Device::cc_cell_sums, 0, (levels)*sizeof(double)));
  }
  // TODO: Return copy of COSQ::q_points
  return nullptr;
  free(cc_sums_lt32);
  free(cc_cardinal_lt32);
}
