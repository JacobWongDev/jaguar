#include "hip/hip_runtime.h"
#include "test_cosq.h"
#include <stdlib.h>
#include <float.h>
#include "spdlog/spdlog.h"
#include "../cuda/nnc.cuh"
#include "../cuda/cc.cuh"
#include "../cuda/dist.cuh"
#include "../cuda/nvidia.cuh"
#include "../ext.h"

#define MAX_ERROR 0.0000001

Split::Split(COSQ* cosq, Device* device) {
  this->cosq = cosq;
  this->device = device;
}

void Split::split_lt32() {
  double* temp = NULL;
  double* s_codebook = (double*) malloc(sizeof(double) * cosq->levels);
  cosq->q_points = (double*) malloc(sizeof(double) * cosq->levels);
  // Compute centroid of training sequence
  double sum = 0;
  for(int i = 0; i < cosq->training_size; i++)
      sum += cosq->training_sequence[i];
  cosq->q_points[0] = sum / cosq->training_size;
  nnc_block_size = {WARP_SIZE, 1, 1};
  cc_cell_sums = (double*) malloc(sizeof(double) * cosq->levels);
  cc_cardinality = (unsigned int*) malloc(sizeof(unsigned int) * cosq->levels);
  unsigned int rate = 0;
  unsigned int s_levels = 1;
  while(s_levels < cosq->levels) {
    // printArr(cosq->q_points, s_levels);
    for(int i = 0; i < s_levels; i++) {
      s_codebook[2*i] = cosq->q_points[i] - delta;
      s_codebook[2*i+1] = cosq->q_points[i] + delta;
    }
    temp = cosq->q_points;
    cosq->q_points = s_codebook;
    s_codebook = temp;
    s_levels <<= 1;
    rate++;
    checkCudaErrors(hipMemset(device->cc_cardinality, 0, s_levels*sizeof(unsigned int)));
    checkCudaErrors(hipMemset(device->cc_cell_sums, 0, s_levels*sizeof(double)));
    nnc_grid_size = {cosq->training_size * s_levels / WARP_SIZE, 1, 1};
    checkCudaErrors(hipMemcpy(device->q_points, cosq->q_points, sizeof(double) * s_levels, hipMemcpyHostToDevice));
    compute_error_matrix(cosq->error_matrix, s_levels, rate);
    checkCudaErrors(hipMemcpy(device->error_matrix, cosq->error_matrix, sizeof(double) * s_levels * s_levels, hipMemcpyHostToDevice));
    s_nnc_lt32<<<nnc_grid_size, nnc_block_size>>>(s_levels, device->training_sequence, device->q_points,
        device->error_matrix, device->cc_cell_sums, device->cc_cardinality);
    checkCudaErrors(hipMemcpy(cc_cell_sums, device->cc_cell_sums, sizeof(double) * s_levels, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(cc_cardinality, device->cc_cardinality, sizeof(unsigned int) * s_levels, hipMemcpyDeviceToHost));
    cc_lt32(s_levels, cosq->error_matrix, cc_cell_sums, cc_cardinality, cosq->q_points);
  }
  free(s_codebook);
  free(cc_cell_sums);
  free(cc_cardinality);
}

void Split::split_ge32() {
  double* temp = NULL;
  double* s_codebook = (double*) malloc(sizeof(double) * cosq->levels);
  cosq->q_points = (double*) malloc(sizeof(double) * cosq->levels);
  cc_cell_sums = (double*) malloc(sizeof(double) * cosq->levels);
  cc_cardinality = (unsigned int*) malloc(sizeof(unsigned int) * cosq->levels);
  // Compute centroid of training sequence
  double sum = 0;
  for(int i = 0; i < cosq->training_size; i++)
    sum += cosq->training_sequence[i];
  cosq->q_points[0] = sum / cosq->training_size;
  nnc_block_size = {WARP_SIZE, 1, 1};
  cc_block_size = {WARP_SIZE, 1, 1};
  unsigned int rate = 0;
  unsigned int s_levels = 1;
  while(s_levels < 32) {
    for(int i = 0; i < s_levels; i++) {
      s_codebook[2*i] = cosq->q_points[i] - delta;
      s_codebook[2*i+1] = cosq->q_points[i] + delta;
    }
    temp = cosq->q_points;
    cosq->q_points = s_codebook;
    s_codebook = temp;
    s_levels <<= 1;
    rate++;
    checkCudaErrors(hipMemset(device->cc_cardinality, 0, s_levels*sizeof(unsigned int)));
    checkCudaErrors(hipMemset(device->cc_cell_sums, 0, s_levels*sizeof(double)));
    nnc_grid_size = {cosq->training_size * s_levels / WARP_SIZE, 1, 1};
    checkCudaErrors(hipMemcpy(device->q_points, cosq->q_points, sizeof(double) * s_levels, hipMemcpyHostToDevice));
    compute_error_matrix(cosq->error_matrix, s_levels, rate);
    checkCudaErrors(hipMemcpy(device->error_matrix, cosq->error_matrix, sizeof(double) * s_levels * s_levels, hipMemcpyHostToDevice));
    s_nnc_lt32<<<nnc_grid_size, nnc_block_size>>>(s_levels, device->training_sequence, device->q_points,
        device->error_matrix, device->cc_cell_sums, device->cc_cardinality);
    checkCudaErrors(hipMemcpy(cc_cell_sums, device->cc_cell_sums, sizeof(double) * s_levels, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(cc_cardinality, device->cc_cardinality, sizeof(unsigned int) * s_levels, hipMemcpyDeviceToHost));
    cc_lt32(s_levels, cosq->error_matrix, cc_cell_sums, cc_cardinality, cosq->q_points);
  }
  nnc_grid_size = {cosq->training_size, 1, 1};
  while(s_levels < cosq->levels) {
    for(int i = 0; i < s_levels; i++) {
      s_codebook[2*i] = cosq->q_points[i] - delta;
      s_codebook[2*i+1] = cosq->q_points[i] + delta;
    }
    temp = cosq->q_points;
    cosq->q_points = s_codebook;
    s_codebook = temp;
    s_levels <<= 1;
    rate++;
    checkCudaErrors(hipMemset(device->cc_cardinality, 0, s_levels*sizeof(unsigned int)));
    checkCudaErrors(hipMemset(device->cc_cell_sums, 0, s_levels*sizeof(double)));
    nnc_smem_size = 2 * s_levels * sizeof(double);
    checkCudaErrors(hipMemcpy(device->q_points, cosq->q_points, sizeof(double) * s_levels, hipMemcpyHostToDevice));
    compute_error_matrix(cosq->error_matrix, s_levels, rate);
    checkCudaErrors(hipMemcpy(device->error_matrix, cosq->error_matrix, sizeof(double) * s_levels * s_levels, hipMemcpyHostToDevice));
    s_nnc_ge32<<<nnc_grid_size, nnc_block_size, nnc_smem_size>>>(s_levels, device->training_sequence,
        device->q_points, device->error_matrix, device->cc_cell_sums, device->cc_cardinality);
    cc_grid_size = {s_levels, 1, 1};
    cc_ge32<<<cc_grid_size, cc_block_size>>>(s_levels, device->q_points, device->error_matrix,
        device->cc_cell_sums, device->cc_cardinality);
    checkCudaErrors(hipMemcpy(cosq->q_points, device->q_points, sizeof(double) * s_levels, hipMemcpyDeviceToHost));
  }
  free(s_codebook);
  free(cc_cell_sums);
  free(cc_cardinality);
}

/**
 * Allocate memory for device arrays.
 */
Device::Device(COSQ* cosq) {
  // Memory allocation
  checkCudaErrors(hipMalloc((void **) &training_sequence, (cosq->training_size)*sizeof(double)));
  checkCudaErrors(hipMalloc((void **) &error_matrix, (cosq->levels)*(cosq->levels)*sizeof(double)));
  checkCudaErrors(hipMalloc((void **) &q_points, (cosq->levels)*sizeof(double)));

  checkCudaErrors(hipMalloc((void **) &q_cells, (cosq->training_size)*sizeof(unsigned int)));

  checkCudaErrors(hipMalloc((void **) &cc_cardinality, (cosq->levels)*sizeof(unsigned int)));
  checkCudaErrors(hipMalloc((void **) &cc_cell_sums, (cosq->levels)*sizeof(double)));

  checkCudaErrors(hipMalloc((void **) &reduction_sums, (cosq->training_size)*sizeof(double)));

  // Memory copying
  checkCudaErrors(hipMemcpy(training_sequence, cosq->training_sequence,
                            (cosq->training_size)*sizeof(double), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemset(cc_cardinality, 0, (cosq->levels)*sizeof(unsigned int)));
  checkCudaErrors(hipMemset(cc_cell_sums, 0, (cosq->levels)*sizeof(double)));

  // CUDA kernel launch params
  nnc_ge32_grid_size = {cosq->training_size, 1, 1};
  nnc_ge32_block_size = {WARP_SIZE, 1, 1};
  nnc_lt32_grid_size = {cosq->training_size * cosq->levels / WARP_SIZE, 1, 1};
  nnc_lt32_block_size = {WARP_SIZE, 1, 1};
  nnc_smem_size = 2 * cosq->levels * sizeof(double);

  cc_grid_size = {cosq->levels, 1, 1};
  cc_block_size = {WARP_SIZE, 1, 1};

  dist_grid_size = {cosq->training_size / WARP_SIZE, 1, 1};
  dist_block_size = {WARP_SIZE, 1, 1};
  dist_smem_size = cosq->levels * sizeof(double);
}

/**
 * Free all memory on device.
 */
Device::~Device() {
  checkCudaErrors(hipFree(training_sequence));
  checkCudaErrors(hipFree(error_matrix));
  checkCudaErrors(hipFree(q_points));
  checkCudaErrors(hipFree(q_cells));
  checkCudaErrors(hipFree(cc_cardinality));
  checkCudaErrors(hipFree(cc_cell_sums));
  checkCudaErrors(hipFree(reduction_sums));
}

/**
 *
 */
COSQ::COSQ(double* training_sequence, const unsigned int* training_size, const unsigned int* bit_rate) {
  this->bit_rate = *bit_rate;
  this->levels = 1 << *bit_rate;
  this->training_sequence = training_sequence;
  this->training_size = *training_size;
  this->error_matrix = (double*) malloc(levels*levels*sizeof(double));
  if(COSQ::error_matrix == nullptr) {
    spdlog::error("Memory Allocation error: Failed to allocate memory for error_matrix!");
    return;
  }
  device = new Device(this);
}

COSQ::~COSQ() {
  free(error_matrix);
  free(q_points);
  delete device;
}

/**
 *
 */
inline double polya_urn_error(int j, int i, int num_bits) {
  double temp;
  int x = j ^ i;
  int previous;
  if(x & 1 == 1) {
    temp = POLYA_EPSILON;
    previous = 1;
  } else {
    temp = 1 - POLYA_EPSILON;
    previous = 0;
  }
  x >>= 1;
  for(int i = 1; i < num_bits; i++) {
    if(x & 1 == 1) {
      temp *= (POLYA_EPSILON + previous * POLYA_DELTA) / (1 + POLYA_DELTA);
      previous = 1;
    } else {
      temp *= ((1 - POLYA_EPSILON) + (1 - previous)*POLYA_DELTA) / (1 + POLYA_DELTA);
      previous = 0;
    }
    x >>= 1;
  }
  return temp;
}

/**
 * TODO: Use CUDA to accelerate this
 */
void compute_error_matrix(double* error_matrix, unsigned int levels, unsigned int bit_rate) {
  for(int i = 0; i < levels; i++) {
    for(int j = 0; j < levels; j++) {
      error_matrix[j + i * levels] = polya_urn_error(j, i, bit_rate);
    }
  }
}

void cc_lt32(unsigned int levels, double* error_matrix, double* cc_sums,
    unsigned int* cc_cardinality, double* q_points) {
  double numerator = 0;
  double denominator = 0;
  for (int j = 0; j < levels; j++) {
    for (int i = 0; i < levels; i++) {
        numerator += error_matrix[j + levels * i] * cc_sums[i];
    }
    for (int i = 0; i < levels; i++) {
        denominator += error_matrix[j + levels * i] * cc_cardinality[i];
    }
    q_points[j] = numerator / denominator;
    numerator = 0;
    denominator = 0;
  }
}

void s_nnc_cpu(unsigned int training_size, double* training_sequence, double* codebook, int levels, double* error_matrix,
    double* cc_sums, unsigned int* cc_cardinality) {
  double min = __FLT_MAX__;
  int min_index = -1;
  double sum = 0;
  double c = 0;
  for(int i = 0; i < training_size; i++) {
    double target = training_sequence[i];
    for(int l = 0; l < levels; l++) {
      // Kahan summation
      for(int j = 0; j < levels; j++) {
        double y = error_matrix[levels*l + j] * (target - codebook[j]) * (target - codebook[j]) - c;
        double t = sum + y;
        c = (t - sum) - y;
        sum = t;
      }
      if(sum < min) {
        min_index = l;
        min = sum;
      }
      sum=0;
      c=0;
    }
    // For Centroid Condition
    cc_cardinality[min_index]++; // update count
    cc_sums[min_index] += target; // running sum
    sum = 0;
    min_index = -1;
    min = __FLT_MAX__;
  }
}

void nnc_cpu(unsigned int training_size, unsigned int* cells, double* training_sequence, double* codebook, int levels, double* error_matrix,
    double* cell_sums, double* cc_sums, unsigned int* cc_cardinality) {
  double min = __FLT_MAX__;
  int min_index = -1;
  double sum = 0;
  double c = 0;
  for(int i = 0; i < training_size; i++) {
    double target = training_sequence[i];
    for(int l = 0; l < levels; l++) {
      // Kahan summation
      for(int j = 0; j < levels; j++) {
        double y = error_matrix[levels*l + j] * (target - codebook[j]) * (target - codebook[j]) - c;
        double t = sum + y;
        c = (t - sum) - y;
        sum = t;
      }
      if(sum < min) {
        min_index = l;
        min = sum;
      }
      cell_sums[levels*i + l] = sum;
      sum=0;
      c=0;
    }
    cells[i] = min_index;
    // For Centroid Condition
    cc_cardinality[min_index]++; // update count
    cc_sums[min_index] += target; // running sum
    sum = 0;
    min_index = -1;
    min = __FLT_MAX__;
  }
}

void cc_cpu(int levels, double* error_matrix, double* cc_sums, unsigned int* cc_cardinality, double* codebook) {
  double numerator = 0;
  double denominator = 0;
  for (int j = 0; j < levels; j++) {
    for (int i = 0; i < levels; i++) {
        numerator += error_matrix[j + levels * i] * cc_sums[i];
    }
    for (int i = 0; i < levels; i++) {
        denominator += error_matrix[j + levels * i] * cc_cardinality[i];
    }
    codebook[j] = numerator / denominator;
    numerator = 0;
    denominator = 0;
  }
}

double distortion_cpu(unsigned int training_size, unsigned int levels, double* training_sequence, double* error_matrix, double* codebook, unsigned int* cells) {
  double d = 0;
  double c = 0;
  for(int i = 0; i < training_size; i++) {
    for(int j = 0; j < levels; j++) {
      double y = error_matrix[j + levels*cells[i]] * (training_sequence[i] - codebook[j]) * (training_sequence[i] - codebook[j]) - c;
      double t = d + y;
      c = (t - d) - y;
      d = t;
    }
  }
  return d / training_size;
}

void nnc_cc_test(unsigned int training_size, unsigned int levels, double* training_sequence, unsigned int* cpu_cells,
                 unsigned int* cuda_cc_cardinality, double* cuda_cc_training_sums) {
  // First, take cpu cells and compute training sequence sums and count cardinality.
  double cc_training_sums[levels] = {};
  unsigned int cardinality[levels] = {};
  unsigned int idx;
  double c[levels] = {};
  double sum;
  for(int i = 0; i < training_size; i++) {
    idx = cpu_cells[i];
    cardinality[idx]++;
    // Kahan summation on cc_training_sums[idx].
    sum = cc_training_sums[idx];
    double y = training_sequence[i] - c[idx];
    double t = sum + y;
    c[idx] = (t - sum) - y;
    cc_training_sums[idx] = t;
  }
  bool equal = true;
  spdlog::info("Executing test cell cardinality and min sums...");
  for(int i = 0; i < levels; i++) {
    if((cuda_cc_cardinality[i] != cardinality[i]) || abs(cc_training_sums[i] - cuda_cc_training_sums[i]) > MAX_ERROR) {
      spdlog::error("Codebook element: {:d}. Cardinality of CUDA {:d} vs. CPU {:d} ", i, cuda_cc_cardinality[i], cardinality[i]);
      spdlog::error("Codebook element: {:d}. cc sum of CUDA {:f} vs. cc sum CPU {:f} ", i, cuda_cc_training_sums[i], cc_training_sums[i]);
      spdlog::error("NNC CC TEST failed!");
      equal = false;
      break;
    }
  }
  if(equal)
    spdlog::info("NNC CC TEST PASSED");
}

void nnc_cells_test(unsigned int training_size, unsigned int levels, unsigned int* cuda_cells, unsigned int* cpu_cells, double* nnc_sums) {
  bool equal = true;
  spdlog::info("NNC: Executing test on cells...");
  for(int i = 0; i < training_size; i++) {
    if((cuda_cells[i] != cpu_cells[i]) && (nnc_sums[i*levels + cuda_cells[i]] != nnc_sums[i*levels + cpu_cells[i]])) {
      spdlog::error("Training element: {:d}. CUDA {:d} vs. CPU {:d} ", i, cuda_cells[i], cpu_cells[i]);
      spdlog::error("Training element: {:d}. CUDA min sum {:f} vs. CPU min sum {:f} ", i, nnc_sums[i*levels + cuda_cells[i]], nnc_sums[i*levels + cpu_cells[i]]);
      spdlog::info("NNC CELLS TEST FAILED");
      equal = false;
      break;
    }
  }
  if(equal)
    spdlog::info("NNC CELLS TEST PASSED");
}

void cc_correct(double* codebook_seq, double* codebook_cuda, unsigned int levels) {
  spdlog::info("NNC: Performing correctness test CC");
  bool correct = true;
  for (int i = 0; i < levels; i++) {
    if (fabsf64(codebook_seq[i] - codebook_cuda[i]) > MAX_ERROR) {
        spdlog::error("The codebooks DO NOT match!\n");
        spdlog::error("Disagreement at {:d}: codebook_seq {:f}, codebook gpu {:f}", i, codebook_seq[i], codebook_cuda[i]);
        correct = false;
        break;
    }
  }
  if (correct)
    spdlog::info("The codebooks match! CC Correctness test passed!\n");
}

void split_test(double* codebook, double* training_sequence, unsigned int training_size, unsigned int levels) {
  double delta = 0.001;
  double* temp = NULL;
  double* s_codebook = (double*) malloc(sizeof(double) * levels);
  double* codebook_seq = (double*) malloc(sizeof(double) * levels);
  double* cc_cell_sums = (double*) malloc(sizeof(double) * levels);
  double* s_error_matrix = (double*) malloc(sizeof(double) * levels * levels);
  unsigned int* cc_cardinality = (unsigned int*) malloc(sizeof(unsigned int) * levels);
  // Compute centroid of training sequence
  double sum = 0;
  for(int i = 0; i < training_size; i++)
    sum += training_sequence[i];
  codebook_seq[0] = sum / training_size;
  // Splitting loop
  unsigned int rate = 0;
  unsigned int s_levels = 1;
  while(s_levels < levels) {
    for(int i = 0; i < s_levels; i++) {
      s_codebook[2*i] = codebook_seq[i] - delta;
      s_codebook[2*i+1] = codebook_seq[i] + delta;
    }
    temp = codebook_seq;
    codebook_seq = s_codebook;
    s_codebook = temp;
    s_levels <<= 1;
    rate++;
    memset(cc_cell_sums, 0, sizeof(double) * s_levels);
    memset(cc_cardinality, 0, sizeof(unsigned int) * s_levels);
    compute_error_matrix(s_error_matrix, s_levels, rate);
    s_nnc_cpu(training_size, training_sequence, codebook_seq, s_levels, s_error_matrix, cc_cell_sums, cc_cardinality);
    cc_cpu(s_levels, s_error_matrix, cc_cell_sums, cc_cardinality, codebook_seq);
  }
  bool correct = true;
  spdlog::info("Split: Performing correctness test");
  for(int i = 0; i < levels; i++) {
    if (fabsf64(codebook_seq[i] - codebook[i]) > MAX_ERROR) {
      spdlog::error("The split codebooks DO NOT match!\n");
      spdlog::error("Disagreement at {:d}: codebook_seq {:f}, codebook gpu {:f}", i, codebook_seq[i], codebook[i]);
      correct = false;
      break;
    }
  }
  if(correct)
    spdlog::info("The codebooks match! Split Correctness test passed!\n");
  free(cc_cell_sums);
  free(cc_cardinality);
  free(s_error_matrix);
  free(s_codebook);
  free(codebook_seq);
}

/**
 *
 */
void COSQ::cosq_lt32() {
  double dist_prev = DBL_MAX, dist_curr = 0;
  Split split(this, device);
  split.split_lt32();
  split_test(q_points, training_sequence, training_size, levels);
  checkCudaErrors(hipMemcpy(device->q_points, q_points, levels * sizeof(double), hipMemcpyHostToDevice));
  compute_error_matrix(error_matrix, levels, bit_rate);
  checkCudaErrors(hipMemcpy(device->error_matrix, error_matrix, levels * levels * sizeof(double), hipMemcpyHostToDevice));

  // Testing data /////////////////////////////////////////////////////////////////////////////
  unsigned int* cpu_cells = (unsigned int*) malloc(sizeof(unsigned int) * training_size);
  unsigned int* cuda_cells = (unsigned int*) malloc(sizeof(unsigned int) * training_size);
  double* all_sums_nnc = (double*) malloc(sizeof(double) * training_size * levels);
  double* cpu_cc_cell_sums = (double*) malloc(sizeof(double) * levels);
  unsigned int* cpu_cc_cardinal = (unsigned int*) malloc(sizeof(unsigned int) * levels);
  double* cuda_cc_training_sums = (double*) malloc(sizeof(double) * levels);
  unsigned int* cuda_cc_cardinality = (unsigned int*) malloc(sizeof(unsigned int) * levels);
  double* cuda_codebook = (double*) malloc(sizeof(double) * levels);
  memset(cpu_cc_cell_sums, 0, sizeof(double) * levels);
  memset(cpu_cc_cardinal, 0, sizeof(unsigned int) * levels);
  //////////////////////////////////////////////////////////////////////////////////////////////
  // COSQ algorithm
  while(true) {
    checkCudaErrors(hipMemset(device->cc_cardinality, 0, levels*sizeof(unsigned int)));
    checkCudaErrors(hipMemset(device->cc_cell_sums, 0, levels*sizeof(double)));
    // NNC
    nnc_lt32<<<device->nnc_lt32_grid_size, device->nnc_lt32_block_size>>>(levels, device->training_sequence, device->q_points,
        device->error_matrix, device->q_cells, device->cc_cell_sums, device->cc_cardinality);
    nnc_cpu(training_size, cpu_cells, training_sequence, q_points, levels, error_matrix, all_sums_nnc, cpu_cc_cell_sums, cpu_cc_cardinal);
    checkCudaErrors(hipMemcpy(cuda_cells, device->q_cells, sizeof(unsigned int) * training_size, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(cuda_cc_cardinality, device->cc_cardinality, levels*sizeof(unsigned int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(cuda_cc_training_sums, device->cc_cell_sums, levels*sizeof(double), hipMemcpyDeviceToHost));
    nnc_cells_test(training_size, levels, cuda_cells, cpu_cells, all_sums_nnc);
    nnc_cc_test(training_size, levels, training_sequence, cpu_cells, cuda_cc_cardinality, cuda_cc_training_sums);

    // CC
    cc_lt32(levels, error_matrix, cpu_cc_cell_sums, cpu_cc_cardinal, q_points);
    checkCudaErrors(hipMemcpy(device->q_points, q_points, sizeof(double) * levels, hipMemcpyHostToDevice));
    spdlog::info("CC: Skipping test, using sequential impl!");

    // Distortion
    distortion_gather_lt32<<<device->dist_grid_size, device->dist_block_size>>>(levels, device->training_sequence,
        device->q_points, device->error_matrix, device->q_cells, device->reduction_sums);
    dist_curr = distortion_reduce(training_size, device->reduction_sums);
    double d_cpu = distortion_cpu(training_size, levels, training_sequence, error_matrix, q_points, cpu_cells);
    if(fabsf64(d_cpu - dist_curr) > MAX_ERROR) {
      spdlog::error("Distortion test failed! CPU {:f} vs. GPU {:f}", d_cpu, dist_curr);
    } else {
      spdlog::info("Distortion test passed! CPU {:f} vs. GPU {:f}", d_cpu, dist_curr);
    }
    if((dist_prev - dist_curr) / dist_prev < THRESHOLD) {
      break;
    }
    dist_prev = dist_curr;
    memset(cpu_cc_cell_sums, 0, sizeof(double) * levels);
    memset(cpu_cc_cardinal, 0, sizeof(unsigned int) * levels);
  }
  free(cpu_cells);
  free(cuda_cells);
  free(cuda_codebook);
  free(all_sums_nnc);
  free(cpu_cc_cell_sums);
  free(cpu_cc_cardinal);
  free(cuda_cc_training_sums);
  free(cuda_cc_cardinality);
}

void COSQ::cosq_ge32() {
  double dist_prev = DBL_MAX, dist_curr = 0;
  Split split(this, device);
  split.split_ge32();
  split_test(q_points, training_sequence, training_size, levels);
  checkCudaErrors(hipMemcpy(device->q_points, q_points, levels * sizeof(double), hipMemcpyHostToDevice));
  compute_error_matrix(error_matrix, levels, bit_rate);
  checkCudaErrors(hipMemcpy(device->error_matrix, error_matrix, levels * levels * sizeof(double), hipMemcpyHostToDevice));

  // Testing data /////////////////////////////////////////////////////////////////////////////
  unsigned int* cpu_cells = (unsigned int*) malloc(sizeof(unsigned int) * training_size);
  unsigned int* cuda_cells = (unsigned int*) malloc(sizeof(unsigned int) * training_size);
  double* all_sums_nnc = (double*) malloc(sizeof(double) * training_size * levels);
  double* cpu_cc_cell_sums = (double*) malloc(sizeof(double) * levels);
  unsigned int* cpu_cc_cardinal = (unsigned int*) malloc(sizeof(unsigned int) * levels);
  double* cuda_cc_training_sums = (double*) malloc(sizeof(double) * levels);
  unsigned int* cuda_cc_cardinality = (unsigned int*) malloc(sizeof(unsigned int) * levels);
  double* cuda_codebook = (double*) malloc(sizeof(double) * levels);
  memset(cpu_cc_cell_sums, 0, sizeof(double) * levels);
  memset(cpu_cc_cardinal, 0, sizeof(unsigned int) * levels);
  //////////////////////////////////////////////////////////////////////////////////////////////
  // COSQ algorithm
  while(true) {
    checkCudaErrors(hipMemset(device->cc_cardinality, 0, levels*sizeof(unsigned int)));
    checkCudaErrors(hipMemset(device->cc_cell_sums, 0, levels*sizeof(double)));
    // NNC
    nnc_ge32<<<device->nnc_ge32_grid_size, device->nnc_ge32_block_size, device->nnc_smem_size>>>(levels, device->training_sequence, device->q_points,
        device->error_matrix, device->q_cells, device->cc_cell_sums, device->cc_cardinality);
    nnc_cpu(training_size, cpu_cells, training_sequence, q_points, levels, error_matrix, all_sums_nnc, cpu_cc_cell_sums, cpu_cc_cardinal);
    checkCudaErrors(hipMemcpy(cuda_cells, device->q_cells, training_size*sizeof(unsigned int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(cuda_cc_cardinality, device->cc_cardinality, levels*sizeof(unsigned int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(cuda_cc_training_sums, device->cc_cell_sums, levels*sizeof(double), hipMemcpyDeviceToHost));
    nnc_cells_test(training_size, levels, cuda_cells, cpu_cells, all_sums_nnc);
    nnc_cc_test(training_size, levels, training_sequence, cpu_cells, cuda_cc_cardinality, cuda_cc_training_sums);
    // CC
    cc_ge32<<<device->cc_grid_size, device->cc_block_size>>>(levels, device->q_points, device->error_matrix,
        device->cc_cell_sums, device->cc_cardinality);
    cc_cpu(levels, error_matrix, cpu_cc_cell_sums, cpu_cc_cardinal, q_points);
    checkCudaErrors(hipMemcpy(cuda_codebook, device->q_points, levels*sizeof(double), hipMemcpyDeviceToHost));
    cc_correct(q_points, cuda_codebook, levels);
    // Distortion
    distortion_gather_ge32<<<device->dist_grid_size, device->dist_block_size, device->dist_smem_size>>>(levels, device->training_sequence,
        device->q_points, device->error_matrix, device->q_cells, device->reduction_sums);
    dist_curr = distortion_reduce(training_size, device->reduction_sums);
    double d_cpu = distortion_cpu(training_size, levels, training_sequence, error_matrix, q_points, cpu_cells);
    if(fabsf64(d_cpu - dist_curr) > MAX_ERROR) {
      spdlog::error("Distortion test failed! CPU {:f} vs. GPU {:f}", d_cpu, dist_curr);
    } else {
      spdlog::info("Distortion test passed! CPU {:f} vs. GPU {:f}", d_cpu, dist_curr);
    }
    if((dist_prev - dist_curr) / dist_prev < THRESHOLD) {
      break;
    }
    dist_prev = dist_curr;
    memset(cpu_cc_cell_sums, 0, sizeof(double) * levels);
    memset(cpu_cc_cardinal, 0, sizeof(unsigned int) * levels);
  }
  free(cpu_cells);
  free(cuda_cells);
  free(cuda_codebook);
  free(all_sums_nnc);
  free(cpu_cc_cell_sums);
  free(cpu_cc_cardinal);
  free(cuda_cc_training_sums);
  free(cuda_cc_cardinality);
}

/**
 *
 */
void COSQ::train() {
  if(training_sequence == nullptr || training_size == 0) {
    spdlog::error("Failed to train COSQ: Invalid training sequence or size!");
  }
  if(levels >= 32) {
    cosq_ge32();
  } else {
    cosq_lt32();
  }
}
