#include "ext.h"

#define MIN(a, b) ((a < b) ? a : b)

unsigned int nextPow2(unsigned int x) {
  --x;
  x |= x >> 1;
  x |= x >> 2;
  x |= x >> 4;
  x |= x >> 8;
  x |= x >> 16;
  return ++x;
}

bool isPow2(unsigned int x) {
  return ((x & (x - 1)) == 0);
}


////////////////////////////////////////////////////////////////////////////////
// Compute the number of threads and blocks to use for the given reduction
// kernel For the kernels >= 3, we set threads / block to the minimum of
// maxThreads and n/2. For kernels < 3, we set to the minimum of maxThreads and
// n.  For kernel 6, we observe the maximum specified number of blocks, because
// each thread in that kernel can process a variable number of elements.
////////////////////////////////////////////////////////////////////////////////
void getNumBlocksAndThreads(int n, int maxBlocks,
                            int maxThreads, int &blocks, int &threads) {
  // get device capability, to avoid block/grid size exceed the upper bound
  hipDeviceProp_t prop;
  int device;
  checkCudaErrors(hipGetDevice(&device));
  checkCudaErrors(hipGetDeviceProperties(&prop, device));
  threads = (n < maxThreads * 2) ? nextPow2((n + 1) / 2) : maxThreads;
  blocks = (n + (threads * 2 - 1)) / (threads * 2);

  if ((double)threads * blocks >
      (double)prop.maxGridSize[0] * prop.maxThreadsPerBlock) {
    spdlog::error("n is too large, please choose a smaller number!");
  }
  if (blocks > prop.maxGridSize[0]) {
    spdlog::error("Grid size <{:d}> exceeds the device capability <{:d}>, set block size as {:d} (original {:d})",
        blocks, prop.maxGridSize[0], threads * 2, threads);
    blocks /= 2;
    threads *= 2;
  }
  blocks = MIN(maxBlocks, blocks);
}

////////////////////////////////////////////////////////////////////////////////
// Wrapper function for kernel launch
////////////////////////////////////////////////////////////////////////////////
void reduce(int size, int threads, int blocks, double *device_seq, double* device_res) {
  dim3 dimBlock(threads, 1, 1);
  dim3 dimGrid(blocks, 1, 1);

  // when there is only one warp per block, we need to allocate two warps
  // worth of shared memory so that we don't index shared memory out of bounds
  int smemSize = (threads <= 32) ? 2 * threads * sizeof(double) : threads * sizeof(double);
  // For reduce7 kernel we require only blockSize/warpSize
  // number of elements in shared memory
  smemSize = ((threads / 32) + 1) * sizeof(double);
  if(isPow2(size)) {
    switch (threads) {
      case 1024:
        reduce7<1024, true>
            <<<dimGrid, dimBlock, smemSize>>>(device_seq, device_res, size);
        break;
      case 512:
        reduce7<512, true>
            <<<dimGrid, dimBlock, smemSize>>>(device_seq, device_res, size);
        break;

      case 256:
        reduce7<256, true>
            <<<dimGrid, dimBlock, smemSize>>>(device_seq, device_res, size);
        break;

      case 128:
        reduce7<128, true>
            <<<dimGrid, dimBlock, smemSize>>>(device_seq, device_res, size);
        break;

      case 64:
        reduce7<64, true>
            <<<dimGrid, dimBlock, smemSize>>>(device_seq, device_res, size);
        break;

      case 32:
        reduce7<32, true>
            <<<dimGrid, dimBlock, smemSize>>>(device_seq, device_res, size);
        break;

      case 16:
        reduce7<16, true>
            <<<dimGrid, dimBlock, smemSize>>>(device_seq, device_res, size);
        break;

      case 8:
        reduce7<8, true>
            <<<dimGrid, dimBlock, smemSize>>>(device_seq, device_res, size);
        break;

      case 4:
        reduce7<4, true>
            <<<dimGrid, dimBlock, smemSize>>>(device_seq, device_res, size);
        break;

      case 2:
        reduce7<2, true>
            <<<dimGrid, dimBlock, smemSize>>>(device_seq, device_res, size);
        break;

      case 1:
        reduce7<1, true>
            <<<dimGrid, dimBlock, smemSize>>>(device_seq, device_res, size);
        break;
    }
  } else {
    switch (threads) {
      case 1024:
        reduce7<1024, false>
            <<<dimGrid, dimBlock, smemSize>>>(device_seq, device_res, size);
        break;
      case 512:
        reduce7<512, false>
            <<<dimGrid, dimBlock, smemSize>>>(device_seq, device_res, size);
        break;

      case 256:
        reduce7<256, false>
            <<<dimGrid, dimBlock, smemSize>>>(device_seq, device_res, size);
        break;

      case 128:
        reduce7<128, false>
            <<<dimGrid, dimBlock, smemSize>>>(device_seq, device_res, size);
        break;

      case 64:
        reduce7<64, false>
            <<<dimGrid, dimBlock, smemSize>>>(device_seq, device_res, size);
        break;

      case 32:
        reduce7<32, false>
            <<<dimGrid, dimBlock, smemSize>>>(device_seq, device_res, size);
        break;

      case 16:
        reduce7<16, false>
            <<<dimGrid, dimBlock, smemSize>>>(device_seq, device_res, size);
        break;

      case 8:
        reduce7<8, false>
            <<<dimGrid, dimBlock, smemSize>>>(device_seq, device_res, size);
        break;

      case 4:
        reduce7<4, false>
            <<<dimGrid, dimBlock, smemSize>>>(device_seq, device_res, size);
        break;

      case 2:
        reduce7<2, false>
            <<<dimGrid, dimBlock, smemSize>>>(device_seq, device_res, size);
        break;

      case 1:
        reduce7<1, false>
            <<<dimGrid, dimBlock, smemSize>>>(device_seq, device_res, size);
        break;
    }
  }
}

double distortion_reduce(unsigned int training_size, double* device_reduce_sums) {
  unsigned int maxThreads = 256;  // number of threads per block
  unsigned int maxBlocks = 64;
  int cpuFinalThreshold = 1;
  bool needReadBack = true;
  int threads, blocks;
  getNumBlocksAndThreads(training_size, maxBlocks, maxThreads, blocks, threads);
  double* device_res;
  double* result = (double*) malloc(sizeof(double) * blocks);
  checkCudaErrors(hipMalloc((void **)&device_res, sizeof(double) * blocks));
  // checkCudaErrors(hipMemcpy(device_res, result, sizeof(double) * blocks, hipMemcpyHostToDevice));

  // Perform GPU reduction
  double* device_intermediate;
  double gpu_res=0;
  checkCudaErrors(hipMalloc((void **)&device_intermediate, sizeof(double) * blocks));

  reduce(training_size, threads, blocks, device_reduce_sums, device_res);

  int s = blocks;
  while(s > cpuFinalThreshold) {
    int threads = 0, blocks = 0;
    getNumBlocksAndThreads(s, maxBlocks, maxThreads, blocks, threads);
    checkCudaErrors(hipMemcpy(device_intermediate, device_res, s * sizeof(double), hipMemcpyDeviceToDevice));
    reduce(s, threads, blocks, device_intermediate, device_res);
    s = (s + (threads * 2 - 1)) / (threads * 2);
  }

  if (s > 1) {
    // copy result from device to host
    checkCudaErrors(hipMemcpy(result, device_res, s * sizeof(double), hipMemcpyDeviceToHost));
    for (int i = 0; i < s; i++) {
      gpu_res += result[i];
    }
    needReadBack = false;
  }

  if (needReadBack) {
    // copy final sum from device to host
    checkCudaErrors(hipMemcpy(&gpu_res, device_res, sizeof(double), hipMemcpyDeviceToHost));
  }
  free(result);
  checkCudaErrors(hipFree(device_res));
  checkCudaErrors(hipFree(device_intermediate));
  return gpu_res / training_size;
}