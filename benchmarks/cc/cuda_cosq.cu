#include "hip/hip_runtime.h"
#include <random>
#include <chrono>
#include <iostream>
#include "cosq.cuh"

#define TRAINING_SIZE 1048576
#define RATE 8
#define POLYA_EPSILON 0.01
#define POLYA_DELTA 0
#define MAX_ERROR 0.0000001
#define ITER 10

void check(hipError_t error, const char* file, int line) {
    if(hipSuccess != error) {
        printf("CUDA error in %s: line %d code=%d(%s): %s\n", file, line, (unsigned int) error, hipGetErrorName(error), hipGetErrorString(error));
    }
}

#define checkCudaErrors(error) check(error, __FILE__, __LINE__);

void cc_cpu(int levels, double* error_matrix, double* cc_sums, unsigned int* cc_cardinality, double* codebook) {
    double numerator = 0;
    double denominator = 0;
    for (int j = 0; j < levels; j++) {
        for (int i = 0; i < levels; i++) {
            numerator += error_matrix[j + levels * i] * cc_sums[i];
        }
        for (int i = 0; i < levels; i++) {
            denominator += error_matrix[j + levels * i] * cc_cardinality[i];
        }
        codebook[j] = numerator / denominator;
        numerator = 0;
        denominator = 0;
    }
}

inline double polya_urn_error(int j, int i, int num_bits) {
    double temp;
    int x = j ^ i;
    int previous;
    if (x & 1 == 1) {
        temp = POLYA_EPSILON;
        previous = 1;
    } else {
        temp = 1 - POLYA_EPSILON;
        previous = 0;
    }
    x >>= 1;
    for (int i = 1; i < num_bits; i++) {
        if (x & 1 == 1) {
            temp *= (POLYA_EPSILON + previous * POLYA_DELTA) / (1 + POLYA_DELTA);
            previous = 1;
        } else {
            temp *= ((1 - POLYA_EPSILON) + (1 - previous) * POLYA_DELTA) / (1 + POLYA_DELTA);
            previous = 0;
        }
        x >>= 1;
    }
    return temp;
}

double* compute_error_matrix(unsigned int levels) {
    double* error_matrix = (double*)malloc(sizeof(double) * levels * levels);
    for (int i = 0; i < levels; i++) {
        for (int j = 0; j < levels; j++) {
            error_matrix[j + i * levels] = polya_urn_error(j, i, RATE);
        }
    }
    return error_matrix;
}

void cc_correct(double* codebook_seq, double* codebook_cuda, unsigned int levels) {
    bool correct = true;
    for (int i = 0; i < levels; i++) {
        if (abs(codebook_seq[i] - codebook_cuda[i]) > MAX_ERROR) {
            printf("The codebooks DO NOT match!\n");
            printf("Disagreement at %d: codebook_seq %f, codebook gpu %f", i, codebook_seq[i], codebook_cuda[i]);
            correct = false;
            break;
        }
    }
    if (correct)
        printf("The codebooks match! CC Correctness test passed!\n");
}

int main(int argc, char **argv) {
    const unsigned int levels = 1 << RATE;
    double *error_matrix = compute_error_matrix(levels);
    double *codebook_seq = (double *)malloc(sizeof(double) * levels);
    double *codebook_cuda = (double *)malloc(sizeof(double) * levels);
    double *cc_training_sums = (double *)calloc(levels, sizeof(double));
    unsigned int *cc_cardinality = (unsigned int *)calloc(levels, sizeof(unsigned int));
    // intialize codebook to first <levels> training samples
    // initialize training_sums and cc_cardinality
    std::default_random_engine rng;
    std::uniform_int_distribution<int> distribution(1, 100);
    rng.seed(31);
    for (int i = 0; i < levels; i++) {
        cc_training_sums[i] = (double)distribution(rng);
        cc_cardinality[i] = (double)distribution(rng);
    }
    /*****************************************************************************************
     * Tests for Centroid Condition
     *****************************************************************************************/
    std::chrono::_V2::system_clock::time_point start, end;
    std::chrono::nanoseconds exec_time;
    int sum = 0;
    /*
      Sequential CC
    */
    std::cout << ":::::::::::: Performance CPU-only code ::::::::::::" << std::endl;
    for(int i = 0; i < ITER; i++) {
        start = std::chrono::high_resolution_clock::now();
        cc_cpu(levels, error_matrix, cc_training_sums, cc_cardinality, codebook_seq);
        end = std::chrono::high_resolution_clock::now();
        exec_time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start);
        if(i == 0) {
            std::cout << "Warm-up time is " << exec_time.count() << "ns." << std::endl;
        } else {
            sum += exec_time.count();
        }
    }
    std::cout << "The average of the remaining exec times is " << sum / (ITER - 1) << "ns." << std::endl;

    /*
        Cuda CC
    */
    double* device_codebook;
    double* device_error_matrix;
    double* device_cc_training_sums;
    unsigned int* device_cc_cardinality;
    checkCudaErrors(hipMalloc((void **) &device_codebook, levels*sizeof(double)));
    checkCudaErrors(hipMalloc((void **) &device_error_matrix, levels*levels*sizeof(double)));
    checkCudaErrors(hipMalloc((void **) &device_cc_training_sums, levels*sizeof(double)));
    checkCudaErrors(hipMalloc((void **) &device_cc_cardinality, levels*sizeof(double)));
    checkCudaErrors(hipMemcpy(device_error_matrix, error_matrix, levels*levels*sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(device_cc_training_sums, cc_training_sums, levels*sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(device_cc_cardinality, cc_cardinality, levels*sizeof(unsigned int), hipMemcpyHostToDevice));
    dim3 grid_size = {levels, 1, 1};
    dim3 block_size = {32, 1, 1};
    sum = 0;
    std::cout << ":::::::::::: Performance GPU-only code ::::::::::::" << std::endl;
    for(int i = 0; i < ITER; i++) {
        start = std::chrono::high_resolution_clock::now();
        cc<<<grid_size, block_size>>>(levels, device_codebook, device_error_matrix, device_cc_training_sums, device_cc_cardinality);
        hipDeviceSynchronize();
        end = std::chrono::high_resolution_clock::now();
        exec_time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start);
        if(i == 0) {
            std::cout << "Warm-up time is " << exec_time.count() << "ns." << std::endl;
        } else {
            sum += exec_time.count();
        }
    }
    std::cout << "The average of the remaining exec times is " << sum / (ITER - 1) << "ns." << std::endl;
    checkCudaErrors(hipMemcpy(codebook_cuda, device_codebook, levels*sizeof(double), hipMemcpyDeviceToHost));
    printf(":::::::::::: Performing correctness test CC ::::::::::::\n");
    cc_correct(codebook_seq, codebook_cuda, levels);
    checkCudaErrors(hipFree(device_codebook));
    checkCudaErrors(hipFree(device_error_matrix));
    checkCudaErrors(hipFree(device_cc_training_sums));
    checkCudaErrors(hipFree(device_cc_cardinality));
    free(codebook_seq);
    free(codebook_cuda);
    free(cc_cardinality);
    free(cc_training_sums);
    free(error_matrix);
}